#include "hip/hip_runtime.h"
#include "kernel_gpu.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>

namespace GPU_KERNEL
{

namespace internal
{

__global__ void add_internal(int *a, int *b, int *c)
{
  *c = *a + *b;
}

};

void add_gpu(int *a, int *b, int *c)
{
  internal::add_internal<<<1,1>>>(a, b, c);
}

};