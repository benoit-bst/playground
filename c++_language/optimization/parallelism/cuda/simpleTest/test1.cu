#include "hip/hip_runtime.h"
#include "main.hpp"
#include <iostream>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

/*
 * Unité de calcul GPU (Kernel)
 */
__global__ void add(int *a, int *b, int *c)
{
  *c = *a + *b;
}

/*
 * compile : nvcc -o test1 test1.cu
 */
int main(int argc, char* argv[])
{
  int nDevices = -1;
  hipGetDeviceCount(&nDevices);
  std::cout << "Nb Device : " << nDevices << std::endl;

  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);
  std::cout << "Device name : " << prop.name << std::endl;

  // Host Var
  int a = 10;
  int b = 10;
  int c = 0;
  int size = sizeof(int);

  //Device copies of a b c
  int *d_a, *d_b, *d_c;

  // Allocation memoire sur device of a, b ,c
  hipMalloc( &d_a, size);
  hipMalloc( &d_b, size);
  hipMalloc( &d_c, size);

  // Copy data
  hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

  // Launch add() on GPU
  add<<<1,1>>>(d_a, d_b, d_c);

  // Copy result back to host
  hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);

  std::cout << "- a " << a << std::endl;
  std::cout << "- b " << b << std::endl;
  std::cout << "- Results " << c << std::endl;

  // cleanup
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  return 0;
}